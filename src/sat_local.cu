//http://stackoverflow.com/questions/22217628/integral-image-or-summed-area-table-of-2d-matrix-using-cuda-c
#include <iostream>
#include <hip/hip_runtime.h>
#include <stdlib.h> 
#include <stdio.h>
#define BLOCK_DIM_X 16
#define BLOCK_DIM_Y 16
using namespace std;

__global__ void sat(int *a, int*b, int* flag,int rowsTotal,int colsTotal)
{
    // Thread Ids equal to block Ids because the each blocks contains one thread only.
    int col=blockDim.x*blockIdx.x+threadIdx.x;
    int row=blockDim.y*blockIdx.y+threadIdx.y;
        
    if (row>=rowsTotal || row <0) return;
    if (col>=colsTotal || col <0) return;

    int idx = threadIdx.y*blockDim.x+threadIdx.x; // id in block 
    int didx = row*colsTotal+col;  // compute data id
    __shared__ int s[BLOCK_DIM_X*BLOCK_DIM_Y];
    
    s[idx]=0;
    __syncthreads();

    //printf("run kernel\n");      
    
    while (s[BLOCK_DIM_X*BLOCK_DIM_Y-1]==0){
    if (s[idx]==0){
    if (threadIdx.x>0 && threadIdx.y==0)
    {
        if (s[idx-1])
        {
           if (row>0 )
              b[didx]=b[didx-colsTotal]+a[didx]+b[didx-1]-b[didx-colsTotal-1];
           else 
              b[didx]=a[didx]+b[didx-1];
           s[idx]=1;
        }
            
    }

    if (threadIdx.y>0 && threadIdx.x==0)
    {
        if (s[idx-blockDim.x])
        {
           if (col>0 )
              b[didx]=b[didx-colsTotal]+a[didx]+b[didx-1]-b[didx-colsTotal-1];
           else
              b[didx]=a[didx]+b[didx-colsTotal];
           s[idx]=1;
        }

    }
   
    if (threadIdx.y>0 && threadIdx.x>0)
    {
        if (s[idx-blockDim.x] && s[idx-1])
        {
           b[didx]=b[didx-colsTotal]+a[didx]+b[didx-1]-b[didx-colsTotal-1];
           s[idx]=1;
        }

    }
     if (threadIdx.x==0 && threadIdx.y==0)
     {  
       bool ready=false;
       if (blockIdx.x==0&&blockIdx.y==0)
            ready=true;
       if (blockIdx.x==0&&blockIdx.y>0)
       {
          if(flag[(blockIdx.y-1)*colsTotal/blockDim.x])
            ready=true;
       }
       if (blockIdx.x>0&&blockIdx.y==0)
       {
          if(flag[blockIdx.x-1])
            ready=true;
       }
       if (blockIdx.x>0&&blockIdx.y>0)
       {
          if(flag[blockIdx.y*colsTotal/blockDim.x+blockIdx.x-1] && flag[(blockIdx.y-1)*colsTotal/blockDim.x+blockIdx.x])
            ready=true;
       }
       if (ready)
       { 
          if (row>0 && col>0)
              b[didx]=b[didx-colsTotal]+a[didx]+b[didx-1]-b[didx-colsTotal-1];
          if (row==0 && col>0)
              b[didx]=a[didx]+b[didx-1];    
          if (row>0 && col==0)
             b[didx]=b[didx-colsTotal]+a[didx];
          if (row==0 && col==0)
              b[didx]=a[didx];
          s[idx]=1; 
       }
     }

    }
        __syncthreads();
    } 
    atomicExch(flag+blockIdx.y*colsTotal/blockDim.x+blockIdx.x,1);
    // for older version then cuda 3.0, comment out the above line and chnage it to the following.
    // flag[blockIdx.y*colsTotal/blockDim.x+blockIdx.x]=1;
    
}

void cpu_sat(int* a, int* b, int M, int N){

    for(int r=0;r<M;r++)
    {
        for(int c=0; c<N;c++)
        {
            if(r==0) 
            {
              if (c>0)
                  b[r*N+c]=b[r*N+c-1]+a[r*N+c];
              else
                  b[r*N+c]=a[r*N+c];
            }
            else{
              if (c>0)
                  b[r*N+c]=b[r*N+c-1]+a[r*N+c]+b[(r-1)*N+c]-b[(r-1)*N+c-1];
              else
                  b[r*N+c]=a[r*N+c]+b[(r-1)*N+c];
            }
        }
    }

}
int main()
{
    //M is number of rows
    //N is number of columns
    //M,N have to be multiples of BLOCK_DIM_X and BLOCK_DIM_Y
    int M=64,N=64;
    int total_e=M*N;
    int widthstep=total_e*sizeof(int);

    int * matrix_a= (int *)malloc(widthstep);
    int * matrix_b= (int *)malloc(widthstep);
    int * cpu_result = (int *)malloc(widthstep);
    int * h_flag = (int *)malloc(M*N/BLOCK_DIM_X/BLOCK_DIM_Y*sizeof(int));
    //cout<<"Enter elements for "<< M<<"x"<<N<<" matrix";

    for(int r=0;r<M;r++)
    {
        for(int c=0; c<N;c++)
        {
            //cout<<"Enter Matrix element [ "<<r<<","<<c<<"]";
            matrix_a[r*N+c]=rand()%100;
            matrix_b[r*N+c]=0;
        }
 
    }

    for (int i=0;i<M*N/BLOCK_DIM_X/BLOCK_DIM_Y;i++)
    {
        h_flag[i]=0;
    }
    cpu_sat(matrix_a,cpu_result,M,N);
    int * d_matrix_a, * d_matrix_b, * d_flag;

    //cout<<"start copy"<<endl;
    /*
    for(int r=0;r<M;r++)
    {
        for(int c=0; c<N;c++)
        {
            cout << matrix_a[r*N+c]<<" ";
        }
        cout << endl;
    }

    cout<<endl;
    */
    hipMalloc(&d_matrix_a,widthstep);
    hipMalloc(&d_matrix_b,widthstep);
    hipMalloc(&d_flag,M*N/BLOCK_DIM_X/BLOCK_DIM_Y*sizeof(int));
    hipMemcpy(d_matrix_a,matrix_a,widthstep,hipMemcpyHostToDevice);
    hipMemcpy(d_matrix_b,matrix_b,widthstep,hipMemcpyHostToDevice);
    hipMemcpy(d_flag,h_flag,M*N/BLOCK_DIM_X/BLOCK_DIM_Y*sizeof(int),hipMemcpyHostToDevice);

    //Creating a grid where the number of blocks are equal to the number of pixels or input matrix elements.

    //Each block contains only one thread.

    dim3 grid(N/BLOCK_DIM_X,M/BLOCK_DIM_Y);  // grid is two dimensional!!
    dim3 blockdim(BLOCK_DIM_X,BLOCK_DIM_Y);
    sat<<<grid,blockdim>>>(d_matrix_a, d_matrix_b,d_flag,M,N);
    //for (int i=0;i<M/BLOCK_DIM_Y+N/BLOCK_DIM_X-1;i++){
    //    sat<<<grid,blockdim>>>(d_matrix_a, d_matrix_b,M,N,i);
        //cudaThreadSynchronize();
    //}

    //cudaThreadSynchronize();
    hipMemcpy(matrix_b,d_matrix_b,widthstep,hipMemcpyDeviceToHost);
    cout<<"Compare with CPU result: "<<endl;
    int count=0; 
    for(int r=0;r<M;r++)
    {
        for(int c=0; c<N;c++)
        {
            if(cpu_result[r*N+c]!=matrix_b[r*N+c])
            {   
                count+=1;//cout << matrix_b[r*N+c]<<" "<<;
            //    if(r==0)
                cout<<r<<" "<<c<<" cpu:"<<cpu_result[r*N+c]<<" gpu:"<<matrix_b[r*N+c]<<endl;
            }
        }
        //cout << endl;
    }
    cout<<"mismatch: "<<count<<endl;
    //system("pause");

    hipFree(d_matrix_a);
    hipFree(d_matrix_b);
    free(matrix_a);
    free(matrix_b);
    return 0;
}
