#include "hip/hip_runtime.h"
//http://stackoverflow.com/questions/22217628/integral-image-or-summed-area-table-of-2d-matrix-using-cuda-c
#include <iostream>
#include <hip/hip_runtime.h>
#include <stdlib.h> 
#include <stdio.h>
#define BLOCK_DIM_X 16
#define BLOCK_DIM_Y 16
using namespace std;

__global__ void sat(int *a, int*b, int rowsTotal,int colsTotal,int start)
{
    // Thread Ids equal to block Ids because the each blocks contains one thread only.
    int col;
    int row;
    if (start<=rowsTotal/blockDim.y-1)
    {
        row = (colsTotal/blockDim.x-1-blockIdx.x)*blockDim.y+threadIdx.y-blockDim.y*(colsTotal/blockDim.x-1-start);
        col = blockIdx.x*blockDim.x+threadIdx.x;
    }
    else
    {
        row = (colsTotal/blockDim.x-1-blockIdx.x)*blockDim.y+threadIdx.y;
        col = (start-(colsTotal/blockDim.x-1))*blockDim.x+blockIdx.x*blockDim.x+threadIdx.x;
    }
    if (row>=rowsTotal || row <0) return;
    if (col>=colsTotal || col <0) return;

    int idx = threadIdx.y*blockDim.x+threadIdx.x; // id in block 
    int didx = row*colsTotal+col;  // compute data id
    __shared__ int s[BLOCK_DIM_X*BLOCK_DIM_Y];
    
    s[idx]=0;
    __syncthreads();

    //printf("run kernel\n");      
    
    while (s[BLOCK_DIM_X*BLOCK_DIM_Y-1]==0){
    if (s[idx]==0){
    if (threadIdx.x>0 && threadIdx.y==0)
    {
        if (s[idx-1])
        {
           if (row>0 )
              b[didx]=b[didx-colsTotal]+a[didx]+b[didx-1]-b[didx-colsTotal-1];
           else 
              b[didx]=a[didx]+b[didx-1];
           s[idx]=1;
        }
            
    }

    if (threadIdx.y>0 && threadIdx.x==0)
    {
        if (s[idx-blockDim.x])
        {
           if (col>0 )
              b[didx]=b[didx-colsTotal]+a[didx]+b[didx-1]-b[didx-colsTotal-1];
           else
              b[didx]=a[didx]+b[didx-colsTotal];
           s[idx]=1;
        }

    }
   
    if (threadIdx.y>0 && threadIdx.x>0)
    {
        if (s[idx-blockDim.x] && s[idx-1])
        {
           b[didx]=b[didx-colsTotal]+a[didx]+b[didx-1]-b[didx-colsTotal-1];
           s[idx]=1;
        }

    }
     if (threadIdx.x==0 && threadIdx.y==0)
     {   
          if (row>0 && col>0)
              b[didx]=b[didx-colsTotal]+a[didx]+b[didx-1]-b[didx-colsTotal-1];
          if (row==0 && col>0)
              b[didx]=a[didx]+b[didx-1];    
          if (row>0 && col==0)
             b[didx]=b[didx-colsTotal]+a[didx];
          if (row==0 && col==0)
              b[didx]=a[didx];
          s[idx]=1; 
     }

    }
    __syncthreads();
    }
}

void cpu_sat(int* a, int* b, int M, int N){

    for(int r=0;r<M;r++)
    {
        for(int c=0; c<N;c++)
        {
            if(r==0) 
            {
              if (c>0)
                  b[r*N+c]=b[r*N+c-1]+a[r*N+c];
              else
                  b[r*N+c]=a[r*N+c];
            }
            else{
              if (c>0)
                  b[r*N+c]=b[r*N+c-1]+a[r*N+c]+b[(r-1)*N+c]-b[(r-1)*N+c-1];
              else
                  b[r*N+c]=a[r*N+c]+b[(r-1)*N+c];
            }
        }
    }

}
int main()
{
    //M is number of rows
    //N is number of columns
    //M,N have to be multiples of BLOCK_DIM_X and BLOCK_DIM_Y
    int M=256,N=256;
    int total_e=M*N;
    int widthstep=total_e*sizeof(int);

    int * matrix_a= (int *)malloc(widthstep);
    int * matrix_b= (int *)malloc(widthstep);
    int * cpu_result = (int *)malloc(widthstep);
    //cout<<"Enter elements for "<< M<<"x"<<N<<" matrix";

    for(int r=0;r<M;r++)
    {
        for(int c=0; c<N;c++)
        {
            //cout<<"Enter Matrix element [ "<<r<<","<<c<<"]";
            matrix_a[r*N+c]=rand()%100;
            matrix_b[r*N+c]=0;
        }
 
    }
    cpu_sat(matrix_a,cpu_result,M,N);
    int * d_matrix_a, * d_matrix_b;

    //cout<<"start copy"<<endl;
    /*
    for(int r=0;r<M;r++)
    {
        for(int c=0; c<N;c++)
        {
            cout << matrix_a[r*N+c]<<" ";
        }
        cout << endl;
    }

    cout<<endl;
    */
    hipMalloc(&d_matrix_a,widthstep);
    hipMalloc(&d_matrix_b,widthstep);

    hipMemcpy(d_matrix_a,matrix_a,widthstep,hipMemcpyHostToDevice);
    hipMemcpy(d_matrix_b,matrix_b,widthstep,hipMemcpyHostToDevice);

    //Creating a grid where the number of blocks are equal to the number of pixels or input matrix elements.

    //Each block contains only one thread.

    dim3 grid(N/BLOCK_DIM_X);  // grid is one dimensional!!
    dim3 blockdim(BLOCK_DIM_X,BLOCK_DIM_Y);
    for (int i=0;i<M/BLOCK_DIM_Y+N/BLOCK_DIM_X-1;i++){
        sat<<<grid,blockdim>>>(d_matrix_a, d_matrix_b,M,N,i);
        //hipDeviceSynchronize();
    }

    hipDeviceSynchronize();
    hipMemcpy(matrix_b,d_matrix_b,widthstep,hipMemcpyDeviceToHost);
    cout<<"Compare with CPU result: "<<endl;
    int count=0; 
    for(int r=0;r<M;r++)
    {
        for(int c=0; c<N;c++)
        {
            if(cpu_result[r*N+c]!=matrix_b[r*N+c])
            {   
                count+=1;//cout << matrix_b[r*N+c]<<" "<<;
            //    if(r==0)
                cout<<r<<" "<<c<<" cpu:"<<cpu_result[r*N+c]<<" gpu:"<<matrix_b[r*N+c]<<endl;
            }
        }
        //cout << endl;
    }
    cout<<"mismatch: "<<count<<endl;
    //system("pause");

    hipFree(d_matrix_a);
    hipFree(d_matrix_b);
    free(matrix_a);
    free(matrix_b);
    return 0;
}
